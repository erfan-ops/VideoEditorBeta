#include "hip/hip_runtime.h"
#include "effects.cuh"
#include <math_functions.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>


__global__ void censor_kernel(unsigned char* __restrict__ img, const int rows, const int cols, const int pixelWidth, const int pixelHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cols || y >= rows) {
        return;
    }

    int block_y = (y / pixelHeight) * pixelHeight;
    int block_x = (x / pixelWidth) * pixelWidth;

    int blockCenterIdx = ((block_y + pixelHeight / 2) * cols + (block_x + pixelWidth / 2)) * 3; // Top-left pixel index in the block
    int idx = (y * cols + x) * 3;

    for (int c = 0; c < 3; ++c) {
        img[idx + c] = img[blockCenterIdx + c]; // Copy the color from the top-left pixel
    }
}

__global__ void pixelate_kernel(unsigned char* __restrict__ img, int rows, int cols, int pixelWidth, int pixelHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cols || y >= rows) {
        return;
    }

    int blockY = (y / pixelHeight) * pixelHeight;
    int blockX = (x / pixelWidth) * pixelWidth;

    // Calculate the block's end position
    int blockEndX = min(blockX + pixelWidth, cols);
    int blockEndY = min(blockY + pixelHeight, rows);

    // Accumulate the sum of colors in the block
    size_t sum_colors[3] = { 0, 0, 0 };
    int pixelCount = 0;

    for (int y = blockY; y < blockEndY; ++y) {
        for (int x = blockX; x < blockEndX; ++x) {
            int idx = (y * cols + x) * 3;
            for (int c = 0; c < 3; ++c) {
                sum_colors[c] += img[idx + c];
            }
            pixelCount++;
        }
    }

    // Calculate the average color
    unsigned char avg_colors[3];
    for (int c = 0; c < 3; ++c) {
        avg_colors[c] = static_cast<unsigned char>(sum_colors[c] / pixelCount);
    }

    // Apply the average color to the entire block
    for (int y = blockY; y < blockEndY; ++y) {
        for (int x = blockX; x < blockEndX; ++x) {
            int idx = (y * cols + x) * 3;
            for (int c = 0; c < 3; ++c) {
                img[idx + c] = avg_colors[c];
            }
        }
    }
}

__global__ void roundColors_kernel(unsigned char* __restrict__ img, const int nPixels, const int thresh) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int halfThresh = thresh / 2;

    int idx = pIdx * 3;

    int colorValue;
    int result_value;

    for (int c = 0; c < 3; ++c) {
        int cidx = idx + c;
        colorValue = img[cidx] + halfThresh;
        result_value = colorValue - (colorValue % thresh);
        img[cidx] = (result_value < 255) ? result_value : 255;
    }
}

__global__ void horizontalLine_kernel(unsigned char* __restrict__ img, int rows, int cols, int lineWidth, int thresh) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cols || y >= rows || (y % (lineWidth * 2)) >= lineWidth) {
        return;
    }

    int idx = (y * cols + x) * 3;
    int cidx;

    int result_value;

    for (int c = 0; c < 3; ++c) {
        cidx = idx + c;
        result_value = img[cidx] - thresh;
        img[cidx] = (result_value < 0) ? 0 : result_value;
    }
}

__global__ void dynamicColor_kernel(unsigned char* __restrict__ img, const int nPixels, const unsigned char* colors_BGR, const int num_colors) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int idx = pIdx * 3; // Index for the RGB channels
    float mediant = (0.114f * static_cast<float>(img[idx]) +
                     0.587f * static_cast<float>(img[idx + 1]) +
                     0.299f * static_cast<float>(img[idx + 2])) / 255.0f;

    // Calculate the segment of the gradient based on the number of colors
    float segment_size = 1.0f / (num_colors - 1);
    int segment_index = static_cast<int>(mediant / segment_size);
    segment_index = segment_index <= num_colors - 2 ? segment_index : num_colors - 2;

    // Calculate the blending factor within the segment
    float segment_start = segment_index * segment_size;
    float segment_end = (segment_index + 1) * segment_size;
    float scale_factor = (mediant - segment_start) / (segment_end - segment_start);

    // Blend the colors
    for (int i = 0; i < 3; ++i) {
        unsigned char color_start = colors_BGR[segment_index * 3 + i];
        unsigned char color_end = colors_BGR[(segment_index + 1) * 3 + i];
        img[idx + i] = static_cast<unsigned char>(
            color_start + (color_end - color_start) * scale_factor
            );
    }
}

__global__ void nearestColor_kernel(unsigned char* __restrict__ img, const int nPixels, const unsigned char* colors_BGR, const int num_colors) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int idx = pIdx * 3;

    // Get the current pixel's color
    unsigned char b = img[idx];
    unsigned char g = img[idx + 1];
    unsigned char r = img[idx + 2];

    // Initialize variables to find the nearest color
    int min_distance = INT_MAX;
    int nearest_color_idx = 0;

    // Iterate through the palette to find the nearest color
    for (int i = 0; i < num_colors; ++i) {
        int palette_idx = i * 3;

        // Get the palette color
        unsigned char pb = colors_BGR[palette_idx];
        unsigned char pg = colors_BGR[palette_idx + 1];
        unsigned char pr = colors_BGR[palette_idx + 2];

        // Calculate the squared Euclidean distance between the colors
        int db = b - pb;
        int dg = g - pg;
        int dr = r - pr;
        int distance = db * db + dg * dg + dr * dr;

        // Update the nearest color if this one is closer
        if (distance < min_distance) {
            min_distance = distance;
            nearest_color_idx = i;
        }
    }

    // Set the pixel to the nearest color
    int palette_idx = nearest_color_idx * 3;
    img[idx] = colors_BGR[palette_idx];     // Blue
    img[idx + 1] = colors_BGR[palette_idx + 1]; // Green
    img[idx + 2] = colors_BGR[palette_idx + 2]; // Red
}

__global__ void radial_blur_kernel(unsigned char* __restrict__ img, int rows, int cols, float centerX, float centerY, int blurRadius, float intensity) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the thread is within the image bounds
    if (x >= cols || y >= rows) {
        return;
    }

    // Calculate the direction vector from the center to the current pixel
    float dirX = x - centerX;
    float dirY = y - centerY;

    // Normalize the direction vector
    float length = sqrtf(dirX * dirX + dirY * dirY);
    if (length > 0) {
        dirX /= length;
        dirY /= length;
    }

    // Accumulate color values along the radial direction
    float sumR = 0, sumG = 0, sumB = 0;
    int count = 0;

    for (int i = -blurRadius; i <= blurRadius; ++i) {
        int sampleX = x + static_cast<int>(dirX * i * intensity);
        int sampleY = y + static_cast<int>(dirY * i * intensity);

        // Clamp the sample coordinates to the image bounds
        sampleX = max(0, min(sampleX, cols - 1));
        sampleY = max(0, min(sampleY, rows - 1));

        // Get the color at the sampled pixel
        int idx = (sampleY * cols + sampleX) * 3;
        sumR += img[idx];
        sumG += img[idx + 1];
        sumB += img[idx + 2];
        count++;
    }

    // Calculate the average color
    unsigned char avgR = static_cast<unsigned char>(sumR / count);
    unsigned char avgG = static_cast<unsigned char>(sumG / count);
    unsigned char avgB = static_cast<unsigned char>(sumB / count);

    // Write the averaged color back to the image
    int idx = (y * cols + x) * 3;
    img[idx] = avgR;
    img[idx + 1] = avgG;
    img[idx + 2] = avgB;
}

__global__ void reverse_contrast(unsigned char* __restrict__ img, const int nPixels) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pIdx >= nPixels) return;

    int idx = pIdx * 3;

    // Load RGB components and normalize to [0,1]
    float r = img[idx] / 255.0f;
    float g = img[idx + 1] / 255.0f;
    float b = img[idx + 2] / 255.0f;

    // Compute max and min values
    float max_color = fmaxf(fmaxf(r, g), b);
    float min_color = fminf(fminf(r, g), b);

    // Compute original lightness (L)
    float l = 0.5f * (max_color + min_color);

    // Invert lightness
    float inverted_l = 1.0f - l;

    // Avoid division by zero
    float delta = max_color - min_color;
    if (delta < 1e-6f) {
        // If the color is grayscale, simply invert it
        img[idx] = static_cast<unsigned char>(inverted_l * 255.0f);
        img[idx + 1] = static_cast<unsigned char>(inverted_l * 255.0f);
        img[idx + 2] = static_cast<unsigned char>(inverted_l * 255.0f);
        return;
    }

    // Compute saturation (S) to maintain color intensity
    float s = delta / (1.0f - fabsf(2.0f * l - 1.0f));

    // Compute new min/max values based on inverted lightness
    float new_max = inverted_l + s * (1.0f - fabsf(2.0f * inverted_l - 1.0f)) * 0.5f;
    float new_min = inverted_l - s * (1.0f - fabsf(2.0f * inverted_l - 1.0f)) * 0.5f;

    // Remap RGB values while preserving hue
    float new_r = (r == max_color) ? new_max : new_min + (r - min_color) * (new_max - new_min) / delta;
    float new_g = (g == max_color) ? new_max : new_min + (g - min_color) * (new_max - new_min) / delta;
    float new_b = (b == max_color) ? new_max : new_min + (b - min_color) * (new_max - new_min) / delta;

    // Store back to image buffer
    img[idx] = static_cast<unsigned char>(fminf(fmaxf(new_r * 255.0f, 0.0f), 255.0f));
    img[idx + 1] = static_cast<unsigned char>(fminf(fmaxf(new_g * 255.0f, 0.0f), 255.0f));
    img[idx + 2] = static_cast<unsigned char>(fminf(fmaxf(new_b * 255.0f, 0.0f), 255.0f));
}

__device__ static inline void rgb_to_yiq(float r, float g, float b, float& y, float& i, float& q) {
    y = 0.299f * r + 0.587f * g + 0.114f * b;
    i = 0.596f * r - 0.274f * g - 0.322f * b;
    q = 0.211f * r - 0.523f * g + 0.312f * b;
}

__device__ static inline void yiq_to_rgb(float y, float i, float q, float& r, float& g, float& b) {
    r = y + 0.956f * i + 0.621f * q;
    g = y - 0.272f * i - 0.647f * q;
    b = y - 1.106f * i + 1.703f * q;
}

__global__ void shift_hue_kernel(unsigned char* __restrict__ img, const int nPixels, const float rotationFactor) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int idx = pIdx * 3;

    float r = img[idx] / 255.0f;
    float g = img[idx + 1] / 255.0f;
    float b = img[idx + 2] / 255.0f;

    float y, i, q;
    rgb_to_yiq(r, g, b, y, i, q);

    float cos_theta, sin_theta;
    sincospif(rotationFactor, &sin_theta, &cos_theta);

    // Rotate in the I-Q plane
    float new_i = i * cos_theta - q * sin_theta;
    float new_q = i * sin_theta + q * cos_theta;

    yiq_to_rgb(y, new_i, new_q, r, g, b);

    img[idx] = static_cast<unsigned char>(min(max(r, 0.0f), 1.0f) * 255);
    img[idx + 1] = static_cast<unsigned char>(min(max(g, 0.0f), 1.0f) * 255);
    img[idx + 2] = static_cast<unsigned char>(min(max(b, 0.0f), 1.0f) * 255);
}

__global__ void outlines_kernel(unsigned char* __restrict__ img, const unsigned char* __restrict__ img_copy, const int rows, const int cols, const int shiftX, const int shiftY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cols - shiftX || y >= rows - shiftY) return;

    int idx = (y * cols + x) * 3;
    int shiftedIdx = idx + 3 * (shiftY * cols + shiftX);

    for (int c = 0; c < 3; c++) {
        int color_idx = idx + c;
        img[color_idx] = static_cast<unsigned char>(abs(static_cast<short>(img_copy[color_idx]) - img_copy[shiftedIdx + c]));
    }
}

__global__ void subtract_kernel(unsigned char* __restrict__ img1, const unsigned char* __restrict__ img2, const int nPixels) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int idx = pIdx * 3;

    for (int c = 0; c < 3; c++) {
        int color_idx = idx + c;
        short diff = static_cast<short>(img1[color_idx]) - static_cast<short>(img2[color_idx]);
        img1[color_idx] = static_cast<unsigned char>(abs(diff));
    }
}

__global__ void fastBlur_kernel(unsigned char* __restrict__ img, const unsigned char* __restrict__ img_copy, const int rows, const int cols, const int blur_radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the thread is within the image bounds
    if (x >= cols || y >= rows) {
        return;
    }

    int idx = (y * cols + x) * 3;
    int sumR = 0, sumG = 0, sumB = 0;
    int count = 0;

    const int blur_radius_sqr = blur_radius * blur_radius;

    // Iterate over the rounded neighborhood
    for (int i = -blur_radius; i <= blur_radius; ++i) {
        for (int j = -blur_radius; j <= blur_radius; ++j) {
            // Calculate the distance from the center pixel
            float distance_sqr = i * i + j * j;
            if (distance_sqr <= blur_radius_sqr) {
                int sampleX = x + i;
                int sampleY = y + j;

                // Check if the sampled pixel is within the image bounds
                if (sampleX >= 0 && sampleX < cols && sampleY >= 0 && sampleY < rows) {
                    int sampleIdx = (sampleY * cols + sampleX) * 3;
                    sumR += img_copy[sampleIdx];
                    sumG += img_copy[sampleIdx + 1];
                    sumB += img_copy[sampleIdx + 2];
                    count++;
                }
            }
        }
    }

    // Write the averaged color back to the original image
    img[idx] = static_cast<unsigned char>(sumR / count);
    img[idx + 1] = static_cast<unsigned char>(sumG / count);
    img[idx + 2] = static_cast<unsigned char>(sumB / count);
}

__global__ void trueBlur_kernel(unsigned char* __restrict__ img, const unsigned char* __restrict__ img_copy, const int rows, const int cols, const int blur_radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the thread is within the image bounds
    if (x >= cols || y >= rows) {
        return;
    }

    int idx = (y * cols + x) * 3;

    float sumR = 0, sumG = 0, sumB = 0;
    float totalWeight = 0;

    int blur_radius_sqr = blur_radius * blur_radius;

    // Iterate over the rounded neighborhood
    for (int i = -blur_radius - 1; i <= blur_radius + 1; ++i) {
        for (int j = -blur_radius - 1; j <= blur_radius + 1; ++j) {
            int sampleX = x + i;
            int sampleY = y + j;

            // Check if the sampled pixel is within the image bounds
            if (sampleX >= 0 && sampleX < cols && sampleY >= 0 && sampleY < rows) {
                int sampleIdx = (sampleY * cols + sampleX) * 3;

                // Calculate the distance from the center pixel
                float distance_sqr = i * i + j * j;

                // Calculate the blending factor
                float weight = 1.0f;
                if (distance_sqr > blur_radius_sqr) {
                    weight = (blur_radius + 1) - sqrtf(distance_sqr); // Smooth transition beyond the blur radius
                    if (weight < 0) weight = 0; // Clamp to 0 for pixels too far away
                }

                // Accumulate the weighted color values
                sumR += img_copy[sampleIdx] * weight;
                sumG += img_copy[sampleIdx + 1] * weight;
                sumB += img_copy[sampleIdx + 2] * weight;
                totalWeight += weight;
            }
        }
    }

    // Normalize the accumulated color values by the total weight
    if (totalWeight > 0) {
        img[idx] = static_cast<unsigned char>(sumR / totalWeight); // Red
        img[idx + 1] = static_cast<unsigned char>(sumG / totalWeight); // Green
        img[idx + 2] = static_cast<unsigned char>(sumB / totalWeight); // Blue
    }
}

__global__ void monoChrome_kernel(unsigned char* __restrict__ img, const int nPixels) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int idx = pIdx * 3;

    unsigned char m = static_cast<unsigned char>(0.114f * static_cast<float>(img[idx]) +
                                                 0.587f * static_cast<float>(img[idx + 1]) +
                                                 0.299f * static_cast<float>(img[idx + 2]));

    img[idx++] = m;
    img[idx++] = m;
    img[idx] = m;
}

__global__ void passColors_kernel(unsigned char* __restrict__ img, const int nPixels, const float* __restrict__ passThreshValues) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int idx = pIdx * 3;

    img[idx] = passThreshValues[0] * img[idx];
    img[idx] = passThreshValues[1] * img[idx + 1];
    img[idx] = passThreshValues[2] * img[idx + 2];
}

__device__ static inline float calculatePixelWeight(const float x, const float y, const float cx, const float cy, const float r, const float precision) {
    // Define the pixel boundaries
    float x0 = x - 0.5f; // Left edge of the pixel
    float x1 = x + 0.5f; // Right edge of the pixel
    float y0 = y - 0.5f; // Bottom edge of the pixel
    float y1 = y + 0.5f; // Top edge of the pixel

    // Clamp the boundaries to the circle
    x0 = fmaxf(x0, cx - r);
    x1 = fminf(x1, cx + r);
    y0 = fmaxf(y0, cy - r);
    y1 = fminf(y1, cy + r);

    // Calculate the weight of the pixel based on its overlap with the circle
    float weight = 0.0f;
    for (float px = x0; px <= x1; px += precision) {
        for (float py = y0; py <= y1; py += precision) {
            float dx = px - cx;
            float dy = py - cy;
            if (dx * dx + dy * dy <= r * r) {
                weight += precision * precision; // Add the weight of the small square
            }
        }
    }

    return weight;
}

__global__ void preciseBlur_kernel(unsigned char* __restrict__ img, const unsigned char* __restrict__ img_copy, const int rows, const int cols, const int blur_radius, const float precision) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= cols || y >= rows) return;

    // Initialize sums for each color channel
    float sumR = 0.0f, sumG = 0.0f, sumB = 0.0f;
    float totalWeight = 0.0f;

    // Iterate over the circular neighborhood
    for (int i = -blur_radius; i <= blur_radius; ++i) {
        for (int j = -blur_radius; j <= blur_radius; ++j) {
            int sampleX = x + i;
            int sampleY = y + j;

            // Check if the sampled pixel is within the image bounds
            if (sampleX >= 0 && sampleX < cols && sampleY >= 0 && sampleY < rows) {
                // Calculate the weight of the sampled pixel
                float weight = calculatePixelWeight(sampleX, sampleY, x, y, blur_radius, precision);

                // Accumulate the weighted color values
                int sampleIdx = (sampleY * cols + sampleX) * 3;
                sumR += img_copy[sampleIdx] * weight;
                sumG += img_copy[sampleIdx + 1] * weight;
                sumB += img_copy[sampleIdx + 2] * weight;
                totalWeight += weight;
            }
        }
    }

    // Normalize the accumulated color values by the total weight
    if (totalWeight > 0.0f) {
        int idx = (y * cols + x) * 3;
        img[idx] = static_cast<unsigned char>(sumR / totalWeight); // Red
        img[idx + 1] = static_cast<unsigned char>(sumG / totalWeight); // Green
        img[idx + 2] = static_cast<unsigned char>(sumB / totalWeight); // Blue
    }
}

__global__ void inverseColors_kernel(unsigned char* __restrict__ img, const int nPixels) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int idx = pIdx * 3;

    img[idx++] = 255ui8 - img[idx];
    img[idx++] = 255ui8 - img[idx];
    img[idx] = 255ui8 - img[idx];
}

__global__ void blackNwhite_kernel(unsigned char* __restrict__ img, const int nPixels, const float middle) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int idx = pIdx * 3;

    float m = 0.114f * static_cast<float>(img[idx]) +
              0.587f * static_cast<float>(img[idx + 1]) +
              0.299f * static_cast<float>(img[idx + 2]);

    unsigned char c = m > middle ? 255 : 0;

    img[idx++] = c;
    img[idx++] = c;
    img[idx] = c;
}

__global__ void generateBinaryNoise(unsigned char* __restrict__ img, const int nPixels, size_t seed) {
    int pIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pIdx >= nPixels) return;

    int idx = pIdx * 3;

    // Use a more unique seed by incorporating blockIdx.x and threadIdx.x
    hiprandState state;
    hiprand_init(seed, pIdx, 0, &state);

    // Generate a random binary value (0 or 255)
    unsigned char c = (hiprand(&state) & 1) * 255;

    // Set all three channels (RGB) to the same value
    img[idx++] = c;
    img[idx++] = c;
    img[idx] = c;
}
