
#include <hip/hip_runtime.h>
//__host__ void videoMonoMask(
//    const std::wstring& inputPath,
//    const std::wstring& outputPath,
//    const unsigned char* colors_BGR,
//    const size_t nColors
//) {
//    // Generate temporary file names
//    std::wstring current_time = std::to_wstring(std::time(nullptr));
//
//    std::wstring video_root = fileUtils::splitextw(inputPath).first;
//    std::wstring output_ext = fileUtils::splitextw(outputPath).second;
//
//    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
//    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";
//
//    // Extract audio
//    extractAudio(inputPath, temp_audio_name);
//
//    Video video(inputPath, temp_video_name);
//    Timer timer;
//
//    unsigned char* d_img;
//    unsigned char* d_colors_BGR;
//
//    size_t color_size = 3ULL * nColors * sizeof(unsigned char);
//
//    // Allocate device memory
//    checkCudaError(cudaMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
//    checkCudaError(cudaMalloc(&d_colors_BGR, color_size), "Failed to allocate device memory for colors");
//
//    checkCudaError(cudaMemcpy(d_colors_BGR, colors_BGR, color_size, cudaMemcpyHostToDevice), "Failed to copy colors to device");
//
//    // Frame buffer pool (preallocated)
//    const int NUM_BUFFERS = nBuffers;
//    std::queue<cv::Mat> bufferPool;
//    for (int i = 0; i < NUM_BUFFERS; i++) {
//        cv::Mat frame(video.getHeight(), video.getWidth(), video.getType());
//        bufferPool.push(frame);
//    }
//
//    std::queue<cv::Mat> frameQueue;
//    std::mutex queueMutex;
//    std::condition_variable queueCV;
//    std::atomic<bool> isProcessing(true);
//
//    std::mutex bufferMutex;
//    std::condition_variable bufferCV;
//
//    // Writer thread function
//    auto writerThread = [&]() {
//        while (true) {
//            cv::Mat frame;
//            {
//                std::unique_lock<std::mutex> lock(queueMutex);
//                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });
//
//                if (!isProcessing && frameQueue.empty()) break;
//
//                frame = frameQueue.front();
//                frameQueue.pop();
//            }
//
//            video.write(frame);
//
//            // Recycle buffer
//            {
//                std::lock_guard<std::mutex> bufferLock(bufferMutex);
//                bufferPool.push(frame);
//                bufferCV.notify_one();
//            }
//        }
//        };
//
//    cudaStream_t stream;
//    checkCudaError(cudaStreamCreate(&stream), "Failed to create stream");
//
//    int blockSize = 1024;
//    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;
//
//    std::thread writer(writerThread);
//
//    timer.start();
//    while (video.getSuccess()) {
//        std::unique_lock<std::mutex> bufferLock(bufferMutex);
//        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });
//
//        cv::Mat frameBuffer = bufferPool.front();
//        bufferPool.pop();
//        bufferLock.unlock();
//
//        cudaMemcpyAsync(d_img, video.getData(), video.getSize(), cudaMemcpyHostToDevice, stream);
//
//        // fix intelisense
//        dynamicColor_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, video.getNumPixels(), d_colors_BGR, nColors);
//
//        cudaMemcpyAsync(frameBuffer.data, d_img, video.getSize(), cudaMemcpyDeviceToHost, stream);
//        cudaStreamSynchronize(stream);
//
//        {
//            std::lock_guard<std::mutex> frameLock(queueMutex);
//            frameQueue.push(frameBuffer);
//        }
//        queueCV.notify_one();
//
//        timer.update();
//        videoShowProgress(video, timer);
//        video.nextFrame();
//    }
//
//    isProcessing = false;
//    queueCV.notify_one();
//    writer.join();
//
//    // clean up
//    video.release();
//    cudaFree(d_img);
//    cudaStreamDestroy(stream);
//
//    mergeAudio(temp_video_name, temp_audio_name, outputPath);
//
//    fileUtils::deleteFile(temp_video_name);
//    fileUtils::deleteFile(temp_audio_name);
//}
//
//__host__ void videoPassColors(
//    const std::wstring& inputPath,
//    const std::wstring& outputPath,
//    const float* __restrict passThreshValues
//) {
//    // Generate temporary file names
//    std::wstring current_time = std::to_wstring(std::time(nullptr));
//
//    std::wstring video_root = fileUtils::splitextw(inputPath).first;
//    std::wstring output_ext = fileUtils::splitextw(outputPath).second;
//
//    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
//    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";
//
//    // Extract audio
//    extractAudio(inputPath, temp_audio_name);
//
//    Video video(inputPath, temp_video_name);
//    Timer timer;
//
//    unsigned char* d_img;
//    float* d_passThreshValues;
//
//    static constexpr size_t color_size = 3ULL * sizeof(float);
//
//    // Allocate device memory
//    checkCudaError(cudaMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
//    checkCudaError(cudaMalloc(&d_passThreshValues, color_size), "Failed to allocate device memory for colors");
//
//    checkCudaError(cudaMemcpy(d_passThreshValues, passThreshValues, color_size, cudaMemcpyHostToDevice), "Failed to copy colors to device");
//
//    // Frame buffer pool (preallocated)
//    const int NUM_BUFFERS = nBuffers;
//    std::queue<cv::Mat> bufferPool;
//    for (int i = 0; i < NUM_BUFFERS; i++) {
//        cv::Mat frame(video.getHeight(), video.getWidth(), video.getType());
//        bufferPool.push(frame);
//    }
//
//    std::queue<cv::Mat> frameQueue;
//    std::mutex queueMutex;
//    std::condition_variable queueCV;
//    std::atomic<bool> isProcessing(true);
//
//    std::mutex bufferMutex;
//    std::condition_variable bufferCV;
//
//    // Writer thread function
//    auto writerThread = [&]() {
//        while (true) {
//            cv::Mat frame;
//            {
//                std::unique_lock<std::mutex> lock(queueMutex);
//                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });
//
//                if (!isProcessing && frameQueue.empty()) break;
//
//                frame = frameQueue.front();
//                frameQueue.pop();
//            }
//
//            video.write(frame);
//
//            // Recycle buffer
//            {
//                std::lock_guard<std::mutex> bufferLock(bufferMutex);
//                bufferPool.push(frame);
//                bufferCV.notify_one();
//            }
//        }
//        };
//
//    cudaStream_t stream;
//    checkCudaError(cudaStreamCreate(&stream), "Failed to create stream");
//
//    int blockSize = 1024;
//    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;
//
//    std::thread writer(writerThread);
//
//    timer.start();
//    while (video.getSuccess()) {
//        std::unique_lock<std::mutex> bufferLock(bufferMutex);
//        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });
//
//        cv::Mat frameBuffer = bufferPool.front();
//        bufferPool.pop();
//        bufferLock.unlock();
//
//        cudaMemcpyAsync(d_img, video.getData(), video.getSize(), cudaMemcpyHostToDevice, stream);
//
//        // fix intelisense
//        passColors_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, video.getNumPixels(), d_passThreshValues);
//
//        cudaMemcpyAsync(frameBuffer.data, d_img, video.getSize(), cudaMemcpyDeviceToHost, stream);
//        cudaStreamSynchronize(stream);
//
//        {
//            std::lock_guard<std::mutex> frameLock(queueMutex);
//            frameQueue.push(frameBuffer);
//        }
//        queueCV.notify_one();
//
//        timer.update();
//        videoShowProgress(video, timer);
//        video.nextFrame();
//    }
//
//    isProcessing = false;
//    queueCV.notify_one();
//    writer.join();
//
//    // clean up
//    video.release();
//    cudaFree(d_img);
//    cudaStreamDestroy(stream);
//
//    mergeAudio(temp_video_name, temp_audio_name, outputPath);
//
//    fileUtils::deleteFile(temp_video_name);
//    fileUtils::deleteFile(temp_audio_name);
//}
