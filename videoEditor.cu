#pragma once

#include "videoEditor.cuh"
#include "videoEffects.cuh"
#include "utils.h"

#include <Windows.h>


static void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}


__host__ void videoVintage8bit(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned short pixelWidth,
    const unsigned short pixelHeight,
    const unsigned char* colors_BGR,
    const size_t nColors,
    const unsigned char threshold,
    const unsigned short lineWidth,
    const unsigned char lineDarkeningThresh
) {
    // Generate temporary file names
    std::time_t current_time = std::time(nullptr);
    std::wstring time_string = stringUtils::string_to_wstring(std::ctime(&current_time));
    time_string.erase(std::remove(time_string.begin(), time_string.end(), ':'), time_string.end());
    time_string.erase(time_string.find_last_not_of('\n') + 1);

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + time_string + output_ext;
    std::wstring temp_audio_name = video_root + L" " + time_string + L".aac";

    // Extract audio
    std::wstring audio_command = L"ffmpeg -loglevel quiet -i \"" + inputPath + L"\" -vn -acodec copy \"" + temp_audio_name + L"\"";
    execute_command(audio_command);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_colors_BGR;

    size_t color_size = 3ULL * nColors * sizeof(unsigned char);

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_colors_BGR, color_size), "Failed to allocate device memory for colors");

    checkCudaError(hipMemcpy(d_colors_BGR, colors_BGR, color_size, hipMemcpyHostToDevice), "Failed to copy colors to device");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = 4;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        dynamicColor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), d_colors_BGR, nColors);
        censor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), pixelWidth, pixelHeight);
        roundColors_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), threshold);
        horizontalLine_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), lineWidth, lineDarkeningThresh);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    video.release();
    hipFree(d_img);
    hipFree(d_colors_BGR);
    hipStreamDestroy(stream);

    std::wstring merge_command = L"ffmpeg -loglevel quiet -i \"" + temp_video_name + L"\" -i \"" + temp_audio_name + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputPath + L"\" -y";
    execute_command(merge_command);

    execute_command(L"del \"" + temp_video_name + L"\"");
    execute_command(L"del \"" + temp_audio_name + L"\"");
}


__host__ void videoRadialBlur(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    int blurRadius,
    float intensity,
    float centerX,
    float centerY
) {
    // Generate temporary file names
    std::time_t current_time = std::time(nullptr);
    std::wstring time_string = stringUtils::string_to_wstring(std::ctime(&current_time));
    time_string.erase(std::remove(time_string.begin(), time_string.end(), ':'), time_string.end());
    time_string.erase(time_string.find_last_not_of('\n') + 1);

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + time_string + output_ext;
    std::wstring temp_audio_name = video_root + L" " + time_string + L".aac";

    // Extract audio
    std::wstring audio_command = L"ffmpeg -loglevel quiet -i \"" + inputPath + L"\" -vn -acodec copy \"" + temp_audio_name + L"\"";
    execute_command(audio_command);

    Video video(inputPath, temp_video_name);
    Timer timer;

    if (centerX == -1)
        centerX = video.getWidth() / 2.0f;
    if (centerY == -1)
        centerY = video.getHeight() / 2.0f;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = 4;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        radial_blur_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), centerX, centerY, blurRadius, intensity);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    std::wstring merge_command = L"ffmpeg -loglevel quiet -i \"" + temp_video_name + L"\" -i \"" + temp_audio_name + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputPath + L"\" -y";
    execute_command(merge_command);

    execute_command(L"del \"" + temp_video_name + L"\"");
    execute_command(L"del \"" + temp_audio_name + L"\"");
}


__host__ void videoReverseContrast(
    const std::wstring& inputPath,
    const std::wstring& outputPath
) {
    // Generate temporary file names
    std::time_t current_time = std::time(nullptr);
    std::wstring time_string = stringUtils::string_to_wstring(std::ctime(&current_time));
    time_string.erase(std::remove(time_string.begin(), time_string.end(), ':'), time_string.end());
    time_string.erase(time_string.find_last_not_of('\n') + 1);

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + time_string + output_ext;
    std::wstring temp_audio_name = video_root + L" " + time_string + L".aac";

    // Extract audio
    std::wstring audio_command = L"ffmpeg -loglevel quiet -i \"" + inputPath + L"\" -vn -acodec copy \"" + temp_audio_name + L"\"";
    execute_command(audio_command);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = 4;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        reverse_contrast<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth());

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    std::wstring merge_command = L"ffmpeg -loglevel quiet -i \"" + temp_video_name + L"\" -i \"" + temp_audio_name + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputPath + L"\" -y";
    execute_command(merge_command);

    execute_command(L"del \"" + temp_video_name + L"\"");
    execute_command(L"del \"" + temp_audio_name + L"\"");
}


__host__ void videoShiftHue(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    float hue_shift
) {
    // Generate temporary file names
    std::time_t current_time = std::time(nullptr);
    std::wstring time_string = stringUtils::string_to_wstring(std::ctime(&current_time));
    time_string.erase(std::remove(time_string.begin(), time_string.end(), ':'), time_string.end());
    time_string.erase(time_string.find_last_not_of('\n') + 1);

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + time_string + output_ext;
    std::wstring temp_audio_name = video_root + L" " + time_string + L".aac";

    // Extract audio
    std::wstring audio_command = L"ffmpeg -loglevel quiet -i \"" + inputPath + L"\" -vn -acodec copy \"" + temp_audio_name + L"\"";
    execute_command(audio_command);

    Video video(inputPath, temp_video_name);
    Timer timer;

    float rotationFactor = 2.0f * hue_shift;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = 4;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        shift_hue_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), rotationFactor);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    std::wstring merge_command = L"ffmpeg -loglevel quiet -i \"" + temp_video_name + L"\" -i \"" + temp_audio_name + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputPath + L"\" -y";
    execute_command(merge_command);

    execute_command(L"del \"" + temp_video_name + L"\"");
    execute_command(L"del \"" + temp_audio_name + L"\"");
}

