#pragma once

#include "videoEditor.cuh"
#include "videoEffects.cuh"
#include "utils.h"

#include <Windows.h>
#include <filesystem>


constexpr int nBuffers = 8;


static void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}


static void extractAudio(const std::wstring& inputVideo, const std::wstring& outputAudio) {
    std::wstring audio_command = L"ffmpeg -loglevel quiet -i \"" + inputVideo + L"\" -vn -acodec copy \"" + outputAudio + L"\"";
    execute_command(audio_command);
}

static void mergeAudio(const std::wstring& inputVideo, const std::wstring& inputAudio, const std::wstring& outputVideo) {
    std::wstring merge_command = L"ffmpeg -loglevel quiet -i \"" + inputVideo + L"\" -i \"" + inputAudio + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputVideo + L"\" -y";
    execute_command(merge_command);
}


__host__ void videoVintage8bit(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned short pixelWidth,
    const unsigned short pixelHeight,
    const unsigned char* colors_BGR,
    const size_t nColors,
    const unsigned char threshold,
    const unsigned short lineWidth,
    const unsigned char lineDarkeningThresh
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_colors_BGR;

    size_t color_size = 3ULL * nColors * sizeof(unsigned char);

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_colors_BGR, color_size), "Failed to allocate device memory for colors");

    checkCudaError(hipMemcpy(d_colors_BGR, colors_BGR, color_size, hipMemcpyHostToDevice), "Failed to copy colors to device");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    int flatBlockSize = 1024;
    int flatGridSize = (video.getNumPixels() + flatBlockSize - 1) / flatBlockSize;

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        dynamicColor_kernel<<<flatGridSize, flatBlockSize, 0, stream>>>(d_img, video.getNumPixels(), d_colors_BGR, nColors);
        censor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), pixelWidth, pixelHeight);
        roundColors_kernel<<<flatGridSize, flatBlockSize, 0, stream>>>(d_img, video.getNumPixels(), threshold);
        horizontalLine_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), lineWidth, lineDarkeningThresh);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    video.release();
    hipFree(d_img);
    hipFree(d_colors_BGR);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}


__host__ void videoRadialBlur(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    int blurRadius,
    float intensity,
    float centerX,
    float centerY
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    if (centerX == -1)
        centerX = video.getWidth() / 2.0f;
    if (centerY == -1)
        centerY = video.getHeight() / 2.0f;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        radial_blur_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), centerX, centerY, blurRadius, intensity);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}


__host__ void videoReverseContrast(
    const std::wstring& inputPath,
    const std::wstring& outputPath
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    int blockSize = 1024;
    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        reverse_contrast<<<gridSize, blockSize, 0, stream>>>(d_img, video.getNumPixels());

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}


__host__ void videoShiftHue(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    float hue_shift
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    float rotationFactor = 2.0f * hue_shift;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    int blockSize = 1024;
    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        shift_hue_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, video.getNumPixels(), rotationFactor);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoOutlines(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    int shiftX, int shiftY
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_img_copy;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_img_copy, video.getSize()), "Failed to allocate device memory for image copy");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_img_copy, d_img, video.getSize(), hipMemcpyDeviceToDevice, stream);

        // fix intelisense
        outlines_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, d_img_copy, video.getHeight(), video.getWidth(), shiftX, shiftY);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipFree(d_img_copy);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoHighlightMotion(
    const std::wstring& inputPath,
    const std::wstring& outputPath
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_oldImg;
    unsigned char* d_newImg;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_oldImg, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_newImg, video.getSize()), "Failed to allocate device memory for image copy");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    int blockSize = 1024;
    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;

    std::thread writer(writerThread);

    timer.start();
    hipMemcpyAsync(d_newImg, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);
    video.nextFrame();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_oldImg, d_newImg, video.getSize(), hipMemcpyDeviceToDevice, stream);
        hipMemcpyAsync(d_newImg, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        subtract_kernel<<<gridSize, blockSize, 0, stream>>>(d_oldImg, d_newImg, video.getNumPixels());

        hipMemcpyAsync(frameBuffer.data, d_oldImg, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_oldImg);
    hipFree(d_newImg);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoBlur(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const int blurRadius,
    const int blending,
    const float precision
) {

    using KernelFunction = void (*)(unsigned char* __restrict__ img, const unsigned char* __restrict__ img_copy, const int rows, const int cols, const int blur_radius);
    KernelFunction blur_func = nullptr;
    if (blending == 0) {
        blur_func = &fastBlur_kernel;
    }
    else if (blending == 1) {
        blur_func = &trueBlur_kernel;
    }
    else {
        blur_func = &fastBlur_kernel;
    }

    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_img_copy;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_img_copy, video.getSize()), "Failed to allocate device memory for image copy");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    if (blending == 2) {
        while (video.getSuccess()) {
            std::unique_lock<std::mutex> bufferLock(bufferMutex);
            bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

            cv::Mat frameBuffer = bufferPool.front();
            bufferPool.pop();
            bufferLock.unlock();

            hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(d_img_copy, d_img, video.getSize(), hipMemcpyDeviceToDevice, stream);

            // fix intelisense
            preciseBlur_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, d_img_copy, video.getHeight(), video.getWidth(), blurRadius, precision);

            hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
            hipStreamSynchronize(stream);

            {
                std::lock_guard<std::mutex> frameLock(queueMutex);
                frameQueue.push(frameBuffer);
            }
            queueCV.notify_one();

            timer.update();
            videoShowProgress(video, timer);
            video.nextFrame();
        }
    }
    else {
        while (video.getSuccess()) {
            std::unique_lock<std::mutex> bufferLock(bufferMutex);
            bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

            cv::Mat frameBuffer = bufferPool.front();
            bufferPool.pop();
            bufferLock.unlock();

            hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(d_img_copy, d_img, video.getSize(), hipMemcpyDeviceToDevice, stream);

            // fix intelisense
            trueBlur_kernel << <gridDim, blockDim, 0, stream >> > (d_img, d_img_copy, video.getHeight(), video.getWidth(), blurRadius);

            hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
            hipStreamSynchronize(stream);

            {
                std::lock_guard<std::mutex> frameLock(queueMutex);
                frameQueue.push(frameBuffer);
            }
            queueCV.notify_one();

            timer.update();
            videoShowProgress(video, timer);
            video.nextFrame();
        }
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipFree(d_img_copy);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoTrueOutlines(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const int thresh
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_img_copy;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_img_copy, video.getSize()), "Failed to allocate device memory for image copy");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    int blockSize = 1024;
    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_img_copy, d_img, video.getSize(), hipMemcpyDeviceToDevice, stream);

        // fix intelisense
        fastBlur_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, d_img_copy, video.getHeight(), video.getWidth(), thresh);
        subtract_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, d_img_copy, video.getNumPixels());

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipFree(d_img_copy);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoMonoChrome(
    const std::wstring& inputPath,
    const std::wstring& outputPath
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    int blockSize = 1024;
    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        monoChrome_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, video.getNumPixels());

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoMonoMask(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned char* colors_BGR,
    const size_t nColors
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_colors_BGR;

    size_t color_size = 3ULL * nColors * sizeof(unsigned char);

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_colors_BGR, color_size), "Failed to allocate device memory for colors");

    checkCudaError(hipMemcpy(d_colors_BGR, colors_BGR, color_size, hipMemcpyHostToDevice), "Failed to copy colors to device");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    int blockSize = 1024;
    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        dynamicColor_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, video.getNumPixels(), d_colors_BGR, nColors);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoPassColors(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const float* __restrict passThreshValues
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;
    float* d_passThreshValues;

    static constexpr size_t color_size = 3ULL * sizeof(float);

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_passThreshValues, color_size), "Failed to allocate device memory for colors");

    checkCudaError(hipMemcpy(d_passThreshValues, passThreshValues, color_size, hipMemcpyHostToDevice), "Failed to copy colors to device");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    int blockSize = 1024;
    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        passColors_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, video.getNumPixels(), d_passThreshValues);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoPixelate(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned short pixelWidth,
    const unsigned short pixelHeight
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        pixelate_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), pixelWidth, pixelHeight);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoCensor(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned short pixelWidth,
    const unsigned short pixelHeight
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        censor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), pixelWidth, pixelHeight);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoRoundColors(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned char thresh
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    int blockSize = 1024;
    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        roundColors_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, video.getNumPixels(), thresh);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}

__host__ void videoInverseColors(
    const std::wstring& inputPath,
    const std::wstring& outputPath
) {
    // Generate temporary file names
    std::wstring current_time = std::to_wstring(std::time(nullptr));

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + current_time + output_ext;
    std::wstring temp_audio_name = video_root + L" " + current_time + L".aac";

    // Extract audio
    extractAudio(inputPath, temp_audio_name);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = nBuffers;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    int blockSize = 1024;
    int gridSize = (video.getNumPixels() + blockSize - 1) / blockSize;

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        inverseColors_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, video.getNumPixels());

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    // clean up
    video.release();
    hipFree(d_img);
    hipStreamDestroy(stream);

    mergeAudio(temp_video_name, temp_audio_name, outputPath);

    fileUtils::deleteFile(temp_video_name);
    fileUtils::deleteFile(temp_audio_name);
}
