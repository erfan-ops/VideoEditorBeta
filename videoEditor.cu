#pragma once

#include "videoEditor.cuh"
#include "videoEffects.cuh"
#include "utils.h"

#include <Windows.h>


static void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

static std::string to_utf8(const std::wstring& wstr) {
    if (wstr.empty()) return std::string();
    int size_needed = WideCharToMultiByte(CP_UTF8, 0, &wstr[0], (int)wstr.size(), nullptr, 0, nullptr, nullptr);
    std::string str(size_needed, 0);
    WideCharToMultiByte(CP_UTF8, 0, &wstr[0], (int)wstr.size(), &str[0], size_needed, nullptr, nullptr);
    return str;
}

static std::wstring string_to_wstring(const std::string& str) {
    if (str.empty()) return std::wstring();

    // Get the required size of the wide-character buffer
    int size_needed = MultiByteToWideChar(CP_ACP, 0, str.c_str(), (int)str.size(), nullptr, 0);

    // Allocate the wide-character buffer
    std::wstring wstr(size_needed, 0);

    // Perform the conversion
    MultiByteToWideChar(CP_ACP, 0, str.c_str(), (int)str.size(), &wstr[0], size_needed);

    return wstr;
}

__host__ void videoVintage8bit(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    int pixelWidth,
    int pixelHeight,
    const unsigned char* color_BGR,
    int threshold,
    int lineWidth,
    int lineDarkeningThresh) {

    // Generate temporary file names
    std::time_t current_time = std::time(nullptr);
    std::wstring time_string = string_to_wstring(std::ctime(&current_time));
    time_string.erase(std::remove(time_string.begin(), time_string.end(), ':'), time_string.end());
    time_string.erase(time_string.find_last_not_of('\n') + 1);

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + time_string + output_ext;
    std::wstring temp_audio_name = video_root + L" " + time_string + L".aac";

    // Extract audio
    std::wstring audio_command = L"ffmpeg -loglevel quiet -threads " + std::to_wstring(std::thread::hardware_concurrency()) + L" -i \"" + inputPath + L"\" -vn -acodec copy \"" + temp_audio_name + L"\"";
    execute_command(audio_command);

    std::wcout << L"file: " << inputPath << std::endl;

    Video video(to_utf8(inputPath), to_utf8(temp_video_name));
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_color_BGR;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_color_BGR, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color");

    checkCudaError(hipMemcpy(d_color_BGR, color_BGR, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");

    // Thread-safe queue and synchronization tools
    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            std::unique_lock<std::mutex> lock(queueMutex);
            queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

            if (!isProcessing && frameQueue.empty()) {
                break;
            }

            cv::Mat frame = frameQueue.front();
            frameQueue.pop();
            lock.unlock();

            video.write(frame);
        }
        };

    // Launch the writer thread
    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        // Copy frame data to device
        checkCudaError(hipMemcpy(d_img, video.getImage().data, video.getSize(), hipMemcpyHostToDevice), "Failed to copy frame to device");

        // CUDA kernel processing
        dim3 blockDim(32, 32);
        dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

        monoChrome_kernel<<<gridDim, blockDim>>>(d_img, video.getHeight(), video.getWidth(), d_color_BGR);
        checkCudaError(hipDeviceSynchronize(), "monoChrome_kernel failed");

        censor_kernel<<<gridDim, blockDim>>>(d_img, video.getHeight(), video.getWidth(), pixelWidth, pixelHeight);
        checkCudaError(hipDeviceSynchronize(), "censor_kernel failed");

        roundColors_kernel<<<gridDim, blockDim>>>(d_img, video.getHeight(), video.getWidth(), threshold);
        checkCudaError(hipDeviceSynchronize(), "roundColors_kernel failed");

        horizontalLine_kernel<<<gridDim, blockDim>>>(d_img, video.getHeight(), video.getWidth(), lineWidth, lineDarkeningThresh);
        checkCudaError(hipDeviceSynchronize(), "horizontalLine_kernel failed");

        // Copy processed frame back to host
        checkCudaError(hipMemcpy(video.getImage().data, d_img, video.getSize(), hipMemcpyDeviceToHost),
            "Failed to copy processed frame back to host");

        // Push processed frame to the queue
        {
            std::lock_guard<std::mutex> lock(queueMutex);
            frameQueue.push(video.getImage().clone());
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    std::cout << "\n" << std::endl;

    // Signal writer thread to finish
    isProcessing = false;
    queueCV.notify_one();

    // Wait for the writer thread to finish
    writer.join();

    // Release resources
    video.release();
    hipFree(d_img);
    hipFree(d_color_BGR);

    // Merge audio and video
    std::wstring merge_command = L"ffmpeg -loglevel quiet -threads " + std::to_wstring(std::thread::hardware_concurrency()) + L" -i \"" + temp_video_name + L"\" -i \"" + temp_audio_name + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputPath + L"\" -y";
    execute_command(merge_command);

    // Clean up temporary files
    execute_command(L"del \"" + temp_video_name + L"\"");
    execute_command(L"del \"" + temp_audio_name + L"\"");
    /*std::remove(to_utf8(temp_video_name).c_str());
    std::remove(to_utf8(temp_audio_name).c_str());*/
}

__host__ void videoVintage8bit2(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned short& pixelWidth,
    const unsigned short& pixelHeight,
    const unsigned char* color1,
    const unsigned char* color2,
    const unsigned char* color3,
    const unsigned char& threshold,
    const unsigned short& lineWidth,
    const unsigned char& lineDarkeningThresh
) {

    // Generate temporary file names
    std::time_t current_time = std::time(nullptr);
    std::wstring time_string = string_to_wstring(std::ctime(&current_time));
    time_string.erase(std::remove(time_string.begin(), time_string.end(), ':'), time_string.end());
    time_string.erase(time_string.find_last_not_of('\n') + 1);

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + time_string + output_ext;
    std::wstring temp_audio_name = video_root + L" " + time_string + L".aac";

    // Extract audio
    std::wstring audio_command = L"ffmpeg -loglevel quiet -threads " + std::to_wstring(std::thread::hardware_concurrency()) + L" -i \"" + inputPath + L"\" -vn -acodec copy \"" + temp_audio_name + L"\"";
    execute_command(audio_command);

    std::wcout << L"file: " << inputPath << std::endl;

    Video video(to_utf8(inputPath), to_utf8(temp_video_name));
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_color1;
    unsigned char* d_color2;
    unsigned char* d_color3;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_color1, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color1");
    checkCudaError(hipMalloc(&d_color2, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color2");
    checkCudaError(hipMalloc(&d_color3, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color3");

    unsigned char* h_img;
    checkCudaError(hipHostMalloc(&h_img, video.getSize()), "Failed to allocate host memory for pinned image");

    checkCudaError(hipMemcpy(d_color1, color1, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");
    checkCudaError(hipMemcpy(d_color2, color2, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");
    checkCudaError(hipMemcpy(d_color3, color3, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");

    // Thread-safe queue and synchronization tools
    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            std::unique_lock<std::mutex> lock(queueMutex);
            queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

            if (!isProcessing && frameQueue.empty()) {
                break;
            }

            cv::Mat frame = frameQueue.front();
            frameQueue.pop();
            lock.unlock();

            video.write(frame);
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    // CUDA kernel processing
    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    // Launch the writer thread
    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        // Copy frame into pinned memory
        memcpy(h_img, video.getData(), video.getSize());

        // Asynchronous copy to device
        hipMemcpyAsync(d_img, h_img, video.getSize(), hipMemcpyHostToDevice, stream);

        triColor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), d_color1, d_color2, d_color3);
        censor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), pixelWidth, pixelHeight);
        roundColors_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), threshold);
        horizontalLine_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), lineWidth, lineDarkeningThresh);
        
        hipStreamSynchronize(stream);

        // Asynchronous copy back
        hipMemcpyAsync(video.getData(), d_img, video.getSize(), hipMemcpyDeviceToHost, stream);

        // Push processed frame to the queue
        {
            std::lock_guard<std::mutex> lock(queueMutex);
            frameQueue.push(video.getImage().clone());
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    std::cout << "\n" << std::endl;

    // Signal writer thread to finish
    isProcessing = false;
    queueCV.notify_one();

    // Wait for the writer thread to finish
    writer.join();

    // Release resources
    video.release();
    hipFree(d_img);
    hipFree(d_color1);
    hipFree(d_color2);
    hipFree(d_color3);
    hipHostFree(h_img);
    hipStreamDestroy(stream);

    // Merge audio and video
    std::wstring merge_command = L"ffmpeg -loglevel quiet -threads " + std::to_wstring(std::thread::hardware_concurrency()) + L" -i \"" + temp_video_name + L"\" -i \"" + temp_audio_name + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputPath + L"\" -y";
    execute_command(merge_command);

    // Clean up temporary files
    execute_command(L"del \"" + temp_video_name + L"\"");
    execute_command(L"del \"" + temp_audio_name + L"\"");
}



__host__ void videoVintage8bit3(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned short& pixelWidth,
    const unsigned short& pixelHeight,
    const unsigned char* color1,
    const unsigned char* color2,
    const unsigned char* color3,
    const unsigned char& threshold,
    const unsigned short& lineWidth,
    const unsigned char& lineDarkeningThresh
) {
    // Generate temporary file names
    std::time_t current_time = std::time(nullptr);
    std::wstring time_string = string_to_wstring(std::ctime(&current_time));
    time_string.erase(std::remove(time_string.begin(), time_string.end(), ':'), time_string.end());
    time_string.erase(time_string.find_last_not_of('\n') + 1);

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + time_string + output_ext;
    std::wstring temp_audio_name = video_root + L" " + time_string + L".aac";

    // Extract audio
    // std::wstring audio_command = L"ffmpeg -loglevel quiet -threads " + std::to_wstring(std::thread::hardware_concurrency()) + L" -i \"" + inputPath + L"\" -vn -acodec copy \"" + temp_audio_name + L"\"";
    std::wstring audio_command = L"ffmpeg -loglevel quiet -i \"" + inputPath + L"\" -vn -acodec copy \"" + temp_audio_name + L"\"";
    execute_command(audio_command);

    Video video(to_utf8(inputPath), to_utf8(temp_video_name));
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_color1;
    unsigned char* d_color2;
    unsigned char* d_color3;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_color1, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color1");
    checkCudaError(hipMalloc(&d_color2, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color2");
    checkCudaError(hipMalloc(&d_color3, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color3");

    unsigned char* h_img;
    checkCudaError(hipHostMalloc(&h_img, video.getSize()), "Failed to allocate host memory for pinned image");

    checkCudaError(hipMemcpy(d_color1, color1, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");
    checkCudaError(hipMemcpy(d_color2, color2, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");
    checkCudaError(hipMemcpy(d_color3, color3, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = 8;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        bufferPool.push(cv::Mat(video.getImage().size(), video.getImage().type()));
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            std::unique_lock<std::mutex> lock(queueMutex);
            queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

            if (!isProcessing && frameQueue.empty()) break;

            cv::Mat frame = frameQueue.front();
            frameQueue.pop();
            lock.unlock();

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> poolLock(queueMutex);
                bufferPool.push(frame);
            }
            queueCV.notify_one();
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> lock(queueMutex);
        queueCV.wait(lock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        lock.unlock();

        memcpy(h_img, video.getData(), video.getSize());
        hipMemcpyAsync(d_img, h_img, video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        triColor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), d_color1, d_color2, d_color3);
        censor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), pixelWidth, pixelHeight);
        roundColors_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), threshold);
        horizontalLine_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), lineWidth, lineDarkeningThresh);

        hipStreamSynchronize(stream);
        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    video.release();
    hipFree(d_img);
    hipFree(d_color1);
    hipFree(d_color2);
    hipFree(d_color3);
    hipHostFree(h_img);
    hipStreamDestroy(stream);

    // std::wstring merge_command = L"ffmpeg -loglevel quiet -threads " + std::to_wstring(std::thread::hardware_concurrency()) + L" -i \"" + temp_video_name + L"\" -i \"" + temp_audio_name + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputPath + L"\" -y";
    std::wstring merge_command = L"ffmpeg -loglevel quiet -i \"" + temp_video_name + L"\" -i \"" + temp_audio_name + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputPath + L"\" -y";
    execute_command(merge_command);

    execute_command(L"del \"" + temp_video_name + L"\"");
    execute_command(L"del \"" + temp_audio_name + L"\"");
}

