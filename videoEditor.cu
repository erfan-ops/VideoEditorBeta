#pragma once

#include "videoEditor.cuh"
#include "videoEffects.cuh"
#include "utils.h"

#include <Windows.h>


static void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}


__host__ void videoVintage8bit(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned short& pixelWidth,
    const unsigned short& pixelHeight,
    const unsigned char* color1,
    const unsigned char* color2,
    const unsigned char* color3,
    const unsigned char& threshold,
    const unsigned short& lineWidth,
    const unsigned char& lineDarkeningThresh
) {
    // Generate temporary file names
    std::time_t current_time = std::time(nullptr);
    std::wstring time_string = stringUtils::string_to_wstring(std::ctime(&current_time));
    time_string.erase(std::remove(time_string.begin(), time_string.end(), ':'), time_string.end());
    time_string.erase(time_string.find_last_not_of('\n') + 1);

    std::wstring video_root = fileUtils::splitextw(inputPath).first;
    std::wstring output_ext = fileUtils::splitextw(outputPath).second;

    std::wstring temp_video_name = video_root + L" " + time_string + output_ext;
    std::wstring temp_audio_name = video_root + L" " + time_string + L".aac";

    // Extract audio
    std::wstring audio_command = L"ffmpeg -loglevel quiet -i \"" + inputPath + L"\" -vn -acodec copy \"" + temp_audio_name + L"\"";
    execute_command(audio_command);

    Video video(inputPath, temp_video_name);
    Timer timer;

    unsigned char* d_img;
    unsigned char* d_color1;
    unsigned char* d_color2;
    unsigned char* d_color3;

    // Allocate device memory
    checkCudaError(hipMalloc(&d_img, video.getSize()), "Failed to allocate device memory for image");
    checkCudaError(hipMalloc(&d_color1, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color1");
    checkCudaError(hipMalloc(&d_color2, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color2");
    checkCudaError(hipMalloc(&d_color3, 3 * sizeof(unsigned char)), "Failed to allocate device memory for color3");

    checkCudaError(hipMemcpy(d_color1, color1, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");
    checkCudaError(hipMemcpy(d_color2, color2, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");
    checkCudaError(hipMemcpy(d_color3, color3, 3 * sizeof(unsigned char), hipMemcpyHostToDevice), "Failed to copy color to device");

    // Frame buffer pool (preallocated)
    const int NUM_BUFFERS = 4;
    std::queue<cv::Mat> bufferPool;
    for (int i = 0; i < NUM_BUFFERS; i++) {
        cv::Mat frame(video.getImage().size(), video.getImage().type());
        bufferPool.push(frame);
    }

    std::queue<cv::Mat> frameQueue;
    std::mutex queueMutex;
    std::condition_variable queueCV;
    std::atomic<bool> isProcessing(true);

    std::mutex bufferMutex;
    std::condition_variable bufferCV;

    // Writer thread function
    auto writerThread = [&]() {
        while (true) {
            cv::Mat frame;
            {
                std::unique_lock<std::mutex> lock(queueMutex);
                queueCV.wait(lock, [&]() { return !frameQueue.empty() || !isProcessing; });

                if (!isProcessing && frameQueue.empty()) break;

                frame = frameQueue.front();
                frameQueue.pop();
            }

            video.write(frame);

            // Recycle buffer
            {
                std::lock_guard<std::mutex> bufferLock(bufferMutex);
                bufferPool.push(frame);
                bufferCV.notify_one();
            }
        }
        };

    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "Failed to create stream");

    dim3 blockDim(32, 32);
    dim3 gridDim((video.getWidth() + blockDim.x - 1) / blockDim.x, (video.getHeight() + blockDim.y - 1) / blockDim.y);

    std::thread writer(writerThread);

    timer.start();
    while (video.getSuccess()) {
        std::unique_lock<std::mutex> bufferLock(bufferMutex);
        bufferCV.wait(bufferLock, [&]() { return !bufferPool.empty(); });

        cv::Mat frameBuffer = bufferPool.front();
        bufferPool.pop();
        bufferLock.unlock();

        hipMemcpyAsync(d_img, video.getData(), video.getSize(), hipMemcpyHostToDevice, stream);

        // fix intelisense
        triColor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), d_color1, d_color2, d_color3);
        censor_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), pixelWidth, pixelHeight);
        roundColors_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), threshold);
        horizontalLine_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, video.getHeight(), video.getWidth(), lineWidth, lineDarkeningThresh);

        hipMemcpyAsync(frameBuffer.data, d_img, video.getSize(), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        {
            std::lock_guard<std::mutex> frameLock(queueMutex);
            frameQueue.push(frameBuffer);
        }
        queueCV.notify_one();

        timer.update();
        videoShowProgress(video, timer);
        video.nextFrame();
    }

    isProcessing = false;
    queueCV.notify_one();
    writer.join();

    video.release();
    hipFree(d_img);
    hipFree(d_color1);
    hipFree(d_color2);
    hipFree(d_color3);
    hipStreamDestroy(stream);

    std::wstring merge_command = L"ffmpeg -loglevel quiet -i \"" + temp_video_name + L"\" -i \"" + temp_audio_name + L"\" -c:v copy -c:a copy -map 0:v:0 -map 1:a:0 \"" + outputPath + L"\" -y";
    execute_command(merge_command);

    execute_command(L"del \"" + temp_video_name + L"\"");
    execute_command(L"del \"" + temp_audio_name + L"\"");
}
