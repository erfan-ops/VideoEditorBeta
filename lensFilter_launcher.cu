#include "lensFilter_launcher.cuh"

void lensFilter_CUDA(
	const int gridSize, const int blockSize, const hipStream_t stream,
	unsigned char* __restrict d_img, const int size, const float* __restrict passThreshValues
) {
	passColors_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, size, passThreshValues);
}

void lensFilterRGBA_CUDA(
	const int gridSize, const int blockSize, const hipStream_t stream,
	unsigned char* __restrict d_img, const int size, const float* __restrict passThreshValues
) {
	passColorsRGBA_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, size, passThreshValues);
}
