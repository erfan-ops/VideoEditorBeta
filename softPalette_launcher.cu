#include "softPalette_launcher.cuh"

void softPalette(
	const int gridSize, const int blockSize, const hipStream_t stream,
	unsigned char* __restrict d_img, const int nPixels,
	const unsigned char* __restrict colorsBGR, const int numColors
) {
	blendNearestColors_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, nPixels, colorsBGR, numColors);
}

void softPaletteRGBA(
	const int gridSize, const int blockSize, const hipStream_t stream,
	unsigned char* __restrict d_img, const int nPixels,
	const unsigned char* __restrict colorsRGB, const int numColors
) {
	blendNearestColorsRGBA_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, nPixels, colorsRGB, numColors);
}
