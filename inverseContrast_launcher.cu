#include "inverseContrast_launcher.cuh"

void inverseContrast_CUDA(const int gridSize, const int blockSize, const hipStream_t stream, unsigned char* __restrict d_img, const int nPixels) {
	reverse_contrast<<<gridSize, blockSize, 0, stream>>>(d_img, nPixels);
}

void inverseContrastRGBA_CUDA(const int gridSize, const int blockSize, const hipStream_t stream, unsigned char* __restrict d_img, const int nPixels) {
	reverseContrastRGBA_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, nPixels);
}
