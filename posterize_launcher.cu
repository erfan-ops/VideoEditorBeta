#include "posterize_launcher.cuh"

__host__ void posterize(
	const int gridSize, const int blockSize, const hipStream_t stream,
	unsigned char* __restrict d_img, const int size, const float threshold
) {
	roundColors_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, size, threshold);
}

__host__ void posterizeRGBA(
	const int gridSize, const int blockSize, const hipStream_t stream,
	unsigned char* __restrict d_img, const int size, const float threshold
) {
	roundColors_kernel << <gridSize, blockSize, 0, stream >> > (d_img, size, threshold);
}
