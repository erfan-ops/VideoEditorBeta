#include "monoChrome_launcher.cuh"

void monoChrome_CUDA(const int gridSize, const int blockSize, const hipStream_t stream, unsigned char* __restrict d_img, const int nPixels) {
	monoChrome_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, nPixels);
}

void monoChromeRGBA_CUDA(const int gridSize, const int blockSize, const hipStream_t stream, unsigned char* __restrict d_img, const int nPixels) {
	monoChromeRGBA_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, nPixels);
}
