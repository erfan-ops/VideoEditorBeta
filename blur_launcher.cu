#include "blur_launcher.cuh"

__host__ void blur_CUDA(dim3 gridDim, dim3 blockDim, hipStream_t stream,
	unsigned char* __restrict d_img, const unsigned char* __restrict d_img_copy,
	int width, int height, int blurRadius
) {
	fastBlur_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, d_img_copy, height, width, blurRadius);
}

__host__ void blurRGBA_CUDA(dim3 gridDim, dim3 blockDim, hipStream_t stream,
	unsigned char* __restrict d_img, const unsigned char* __restrict d_img_copy,
	int width, int height, int blurRadius
) {
	fastBlurRGBA_kernel<<<gridDim, blockDim, 0, stream>>>(d_img, d_img_copy, height, width, blurRadius);
}
