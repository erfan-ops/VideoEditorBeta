#include "imageEditor.cuh"
#include "effects.cuh"

#include <hip/hip_runtime.h>
#include "image.h"
#include "utils.h"



__host__ void imagePixelate(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned short pixelWidth,
    const unsigned short pixelHeight
) {
    Image img(inputPath);

    unsigned char* d_img;

    dim3 blockDim(32, 32);
    dim3 gridDim((img.getWidth() + blockDim.x - 1) / blockDim.x, (img.getHeight() + blockDim.y - 1) / blockDim.y);

    hipMalloc(&d_img, img.getSize());
    hipMemcpy(d_img, img.getData(), img.getSize(), hipMemcpyHostToDevice);

    pixelate_kernel<<<gridDim, blockDim>>>(d_img, img.getHeight(), img.getWidth(), pixelWidth, pixelHeight);

    hipMemcpy(img.getData(), d_img, img.getSize(), hipMemcpyDeviceToHost);
    hipFree(d_img);

    img.save(outputPath);
}

__host__ void imageCensor(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned short pixelWidth,
    const unsigned short pixelHeight
) {
    Image img(inputPath);

    unsigned char* d_img;

    dim3 blockDim(32, 32);
    dim3 gridDim((img.getWidth() + blockDim.x - 1) / blockDim.x, (img.getHeight() + blockDim.y - 1) / blockDim.y);

    hipMalloc(&d_img, img.getSize());
    hipMemcpy(d_img, img.getData(), img.getSize(), hipMemcpyHostToDevice);

    censor_kernel<<<gridDim, blockDim>>>(d_img, img.getHeight(), img.getWidth(), pixelWidth, pixelHeight);

    hipMemcpy(img.getData(), d_img, img.getSize(), hipMemcpyDeviceToHost);
    hipFree(d_img);

    img.save(outputPath);
}

__host__ void imageRoundColors(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned char thresh
) {
    Image img(inputPath);

    unsigned char* d_img;

    int blockSize = 1024;
    int gridSize = (img.getNumPixels() + blockSize - 1) / blockSize;

    hipMalloc(&d_img, img.getSize());
    hipMemcpy(d_img, img.getData(), img.getSize(), hipMemcpyHostToDevice);

    roundColors_kernel<<<gridSize, blockSize>>>(d_img, img.getSize(), thresh);

    hipMemcpy(img.getData(), d_img, img.getSize(), hipMemcpyDeviceToHost);
    hipFree(d_img);

    img.save(outputPath);
}

__host__ void imageMonoMask(
    const std::wstring& inputPath,
    const std::wstring& outputPath,
    const unsigned char* colors_BGR,
    const int num_colors
) {
    Image img(inputPath);

    unsigned char* d_img;
    unsigned char* d_colors;

    int blockSize = 1024;
    int gridSize = (img.getNumPixels() + blockSize - 1) / blockSize;
    static const size_t colorsSize = 3ull * num_colors * sizeof(unsigned char);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMallocAsync(&d_img, img.getSize(), stream);
    hipMallocAsync(&d_colors, colorsSize, stream);

    hipMemcpyAsync(d_img, img.getData(), img.getSize(), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_colors, colors_BGR, colorsSize, hipMemcpyHostToDevice, stream);

    dynamicColor_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, img.getSize(), d_colors, num_colors);

    hipMemcpyAsync(img.getData(), d_img, img.getSize(), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    hipFreeAsync(d_img, stream);
    hipFreeAsync(d_colors, stream);
    hipStreamDestroy(stream);

    img.save(outputPath);
}
