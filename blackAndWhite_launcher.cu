#include "hueShift_launcher.cuh"

void blackAndWhite_CUDA(
	const int gridSize, const int blockSize, const hipStream_t stream,
	unsigned char* __restrict d_img, const int nPixels, const float middle
) {
	blackNwhite_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, nPixels, middle);
}

void blackAndWhiteRGBA_CUDA(
	const int gridSize, const int blockSize, const hipStream_t stream,
	unsigned char* __restrict d_img, const int nPixels, const float middle
) {
	blackNwhiteRGBA_kernel<<<gridSize, blockSize, 0, stream>>>(d_img, nPixels, middle);
}
